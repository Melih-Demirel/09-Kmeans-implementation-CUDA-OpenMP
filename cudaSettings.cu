#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    // Get the number of CUDA devices
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    // Iterate over each CUDA device
    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        // Print device properties
        std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "  Maximum threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "  Maximum blocks per grid: " << deviceProp.maxGridSize[0] << std::endl;
    }

    return 0;
}
